#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>

#include "../include/gputils/cuda_utils.hpp"

using namespace std;


static void show_device(int device)
{
    cout << "Device " << device << endl;

    // https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaDeviceProp.html
    hipDeviceProp_t prop;
    CUDA_CALL(hipGetDeviceProperties(&prop, device));

    cout << "    name = " << prop.name << "\n"
	 << "    compute capability = " << prop.major << "." << prop.minor << "\n"
	 << "    multiProcessorCount = " << prop.multiProcessorCount << "\n"
	 << "    clockRate = " << prop.clockRate << " kHZ  = " << (prop.clockRate / 1.0e6) << " GHz [deprecated]\n"
	 << "    l2CacheSize = " << prop.l2CacheSize << " bytes = " << (prop.l2CacheSize / pow(2,20.)) << " MB\n"
	 << "    totalGlobalMem = " << prop.totalGlobalMem << " bytes = " << (prop.totalGlobalMem / pow(2,30.)) << " GB\n"
	 << "    memoryClockRate = " << prop.memoryClockRate << " kHZ [deprecated]\n"
	 << "    memoryBusWidth = " << prop.memoryBusWidth << " bits\n"
	 << "    implied global memory bandwidth = " << (prop.memoryClockRate * double(prop.memoryBusWidth) / 1.0e6 / 4.) << " GB/s\n"   // empirical!
	 << endl;
}


int main(int argc, char **argv)
{
    int ndevices = -1;
    CUDA_CALL(hipGetDeviceCount(&ndevices));

    cout << "Number of devices: " << ndevices << endl;

    for (int device = 0; device < ndevices; device++)
	show_device(device);
    
    return 0;
}
